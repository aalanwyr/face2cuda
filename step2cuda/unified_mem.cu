#include "hip/hip_runtime.h"
// vector-add
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

__global__
void initWith(float num, float *a, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
 
   
  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
 
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);
    
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
    
  size_t threadsPerBlock;
  size_t numberOfBlocks;
    
  numberOfBlocks = 32*numberOfSMs;
  threadsPerBlock = 512;
  printf("grid size%ld, block size%ld\n", numberOfBlocks, threadsPerBlock);
    
  /*pre fetch to gpu device*/
  /*to speed up kernel execution time*/
  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);

  initWith<<<numberOfBlocks,threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks,threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks,threadsPerBlock>>>(0, c, N);
  
   
  hipError_t addVectorsErr;
  hipError_t asyncErr;

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
    
  /*prefetch the data into cpu host*/
  /*to speed up d2h memcopy*/
  hipMemPrefetchAsync(c, size, hipCpuDeviceId);
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}