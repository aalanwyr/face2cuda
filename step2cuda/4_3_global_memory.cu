#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"

void sumArraysCPU(float *A, float *B, float *C, int offset, int N) {
    // cpu 侧的计算也需要带偏移
    for (int i=0, k=offset; k<N; i++,k++) {
        C[i] = A[k] + B[k]; // offset 之前的数值应该不参与计算了
    }
}

__global__ void sumArraysGPU(float *A, float *B, float *C, int offset, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  
    int index = tid + offset;  //添加偏移量 从 gld memory 读取数据带地址偏移 由于带了
    if(index<N) {
        C[tid] = A[index] + B[index];
    }
}
 
int main(int argc, char** argv) {

    int len = 1<<24;
    printf("Vector size:%d\n",len);
    int nBytes = len * sizeof(float);

    int offset = 0;
    if( argc>=2 ) {
        offset = atoi(argv[1]); // get the offset value
    }

    //malloc host buffer
    float *a_h = (float*) malloc(nBytes);
    float *b_h = (float*) malloc(nBytes);
    float *res_h = (float*) malloc(nBytes);
    float *res_gpu = (float*) malloc(nBytes);
    memset(res_h, 0, nBytes);
    memset(res_gpu, 0, nBytes);

    initialData(a_h, len);
    initialData(b_h, len);

    //malloc pinned device buffer
    float *a_d, *b_d, *res_d;
    CHECK(hipHostMalloc((float**) &a_d, nBytes))
    CHECK(hipHostMalloc((float**) &b_d, nBytes))
    CHECK(hipHostMalloc((float**) &res_d, nBytes))
    CHECK(hipMemset(res_d,0,nBytes));

    CHECK(hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d, b_h, nBytes, hipMemcpyHostToDevice));

    dim3 block(1024);
    dim3 grid(len/block.x);

    // creat cudaEvent to collect the timestamp
    hipEvent_t custart, custop;
    hipEventCreate(&custart);
    hipEventCreate(&custop);

    hipEventRecord(custart, 0);
    sumArraysGPU<<<grid,block>>> (a_d, b_d, res_d, offset, len);
    hipEventRecord(custop, 0);
    hipEventSynchronize(custop);

    float CudaElaps;
    hipEventElapsedTime(&CudaElaps, custart, custop);

    sumArraysCPU(a_d, b_d, res_h, offset, len);

    CHECK(hipMemcpy(res_gpu, res_d, nBytes, hipMemcpyDeviceToHost));

    //checkResult(res_h, res_gpu, len);
    printf("GPU Execution %f ms\n", CudaElaps);


    free(a_h);
    free(b_h);
    free(res_h);
    free(res_gpu);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);


    return 0;
}
